#include "hip/hip_runtime.h"
#include <Python.h>

__global__
void calc(double x_start, double y_start, double step_x, double step_y, int iters, int *arr)
{
	int x = threadIdx.x;
	int y = blockIdx.x;

	double c_x = x_start + x * step_x;
	double c_y = y_start - y * step_y;
	double res_x = c_x;
	double res_y = c_y;
	double nextnum_x;
	double nextnum_y;

	int color = 0;

	for (int i = 0; i < iters; i++)
	{
		nextnum_x = res_x * res_x - res_y * res_y + c_x;
		nextnum_y = res_x * res_y * 2 + c_y;

		if (res_x == nextnum_x && res_y == nextnum_y) // Iteration has converged
			break;
		else if (res_x * res_x + res_y * res_y > 4) // Iteration escapes
		{
			int intensity = i * 255.0 / iters;
			color = intensity * 256; // Cool green color
			break;
		}
		else
		{
			res_x = nextnum_x;
			res_y = nextnum_y;
		}
	}

	arr[blockDim.x * y + x] = color;
}

static PyObject *cudaiter(PyObject *self, PyObject *args)
{
	int threads = 256;
	
	/*
	 * Screen is first partitioned in sectors of width (threads).
	 * Each sector is calculated separately and finally combined
	 * and inserted to the shared array.
	 */
	
	double x_start, x_end, y_start, y_end;
	int iters, length_x, length_y;
	PyObject *ret;

	if (!PyArg_ParseTuple(args, "ddddiiiO", &x_start, &x_end,
			&y_start, &y_end, &iters, &length_x, &length_y, &ret))
		return NULL;

	double step_x = (x_end - x_start) / length_x;
	double step_y = (y_start - y_end) / length_y;

	int strides = length_x / threads; // Calculate required amount of sectors and if there's a smaller sector at the end.
	int incomplete_stride = length_x % threads;
	
	int *colors[strides + (incomplete_stride != 0 ? 1 : 0)];
	for (int i = 0; i < strides; i++)
	{
		hipMallocManaged(&colors[i], threads * length_y * sizeof(int));
		calc<<<length_y, threads>>>(x_start + i * threads * step_x, y_start, step_x, step_y, iters, colors[i]);
	}
	if (incomplete_stride != 0)
	{
		hipMallocManaged(&colors[strides], incomplete_stride * length_y * sizeof(int));
		calc<<<length_y, incomplete_stride>>>(x_start + strides * threads * step_x, y_start, step_x, step_y, iters, colors[strides]);
		strides += 1; // update value to make sure combination works correctly
	}
	hipDeviceSynchronize();
	
	int *arrs[strides]; // Create an iterator for each sector that helps with combination
	for (int i = 0; i < strides; i++)
		arrs[i] = &colors[i][0];
	
	for (int row = 0; row < length_y; row++)
	{
		for (int col = 0; col < length_x; col++)
		{
			// Sectors are combined and updated to shared Python array
			PyObject* key = PyLong_FromLong(row * length_x + col);
			PyObject* item = PyLong_FromLong(*arrs[col / threads]);
			
			PyObject_SetItem(ret, key, item);
			Py_DECREF(item);
			Py_DECREF(key);
			
			arrs[col / threads] += 1;
		}
	}
	
	for (int i = 0; i < strides; i++)
		hipFree(colors[i]);	
	
	Py_RETURN_NONE;
}


static PyObject* iterate(PyObject* self, PyObject* args)
{
	double x_start, y_start, x_end, y_end;
	int offset, iters, count, length_x, length_y;
	PyObject* arr;

	if (!PyArg_ParseTuple(args, "iiddddiiiO", &offset, &count, &x_start, &x_end, &y_start, &y_end, &iters, &length_x, &length_y, &arr))
		return NULL;

	double x_dist = x_end - x_start;
	double y_dist = y_start - y_end;

	for (int y = offset; y < offset + count; y++)
	{
		for (int x = 0; x < length_x; x++)
		{
			long double c_x = x_start + x * x_dist / length_x;
			long double c_y = y_start - y * y_dist / length_y;
			long double res_x = c_x;
			long double res_y = c_y;
			long double nextnum_x;
			long double nextnum_y;

			int color = 0;

			for (int i = 0; i < iters; i++)
			{
				nextnum_x = res_x * res_x - res_y * res_y + c_x;
				nextnum_y = res_x * res_y * 2 + c_y;

				if (res_x == nextnum_x && res_y == nextnum_y)
					break;
				else if (res_x * res_x + res_y * res_y > 4)
				{
					int intensity = i * 255 / iters;
					color = intensity; // Cool blue color
					break;
				}
				else
				{
					res_x = nextnum_x;
					res_y = nextnum_y;
				}
			}

			PyObject* key = PyLong_FromLong(x + y*length_x);
			PyObject* item = PyLong_FromLong(color);
			PyObject_SetItem(arr, key, item);
			Py_DECREF(item);
			Py_DECREF(key);
		}
	}
	Py_RETURN_NONE;
}

static PyMethodDef CalcMethods[] = {
	{"cpuiter", iterate, METH_VARARGS,
	 "Calculate iterations"},
	{"cudaiter", cudaiter, METH_VARARGS,
	 "Calculate iterations"},
    {NULL, NULL, 0, NULL}        /* Sentinel */
};

static struct PyModuleDef itercalc = {
   PyModuleDef_HEAD_INIT,
   "Mandelbrot set calculator",   /* name of module */
   NULL, /* module documentation, may be NULL */
   -1,       /* size of per-interpreter state of the module,
                or -1 if the module keeps state in global variables. */
   CalcMethods
};

PyMODINIT_FUNC
PyInit_itercalc(void)
{
    return PyModule_Create(&itercalc);
}
